#include "hip/hip_runtime.h"
#include <sailcu/hello.h>

__global__ void add(int *a, int *b, int *c)
{
    int tid = blockIdx.x;
    if (tid < N) {
        for (auto j = 0; j < N; j++) {
            a[tid] = a[tid] + j;
            b[tid] = b[tid] + j * j;
        }
        c[tid] = a[tid] + b[tid];
    }
}

SAIL_NAMESPACE_BEGIN

void printHello()
{
    // std::cout << "Hello you" << std::endl;
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;

    hipMalloc((void**)&dev_a, N * sizeof(int));
    hipMalloc((void**)&dev_b, N * sizeof(int));
    hipMalloc((void**)&dev_c, N * sizeof(int));

    for (int i = 0; i < N; i++) {
        a[i] = -i;
        b[i] = i * i;
    }

    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    add<<<N, 1>>>(dev_a, dev_b, dev_c);

    hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++) {
        // printf("%d + %d = %d \n", a[i], b[i], c[i]);
    }

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}

SAIL_NAMESPACE_END
