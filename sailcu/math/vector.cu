#include "hip/hip_runtime.h"
#include <sailcu/vector.h>

__global__ void dVectorAdd(double* vec_a, double* vec_b, double* vec_c, size_t _size)
{
    int tid = blockIdx.x;
    if (tid < _size) {
        vec_c[tid] = vec_a[tid] + vec_b[tid];
    }
}

SAIL_NAMESPACE_BEGIN

Vector4dCu Vector4dCu::operator+(const Vector4dCu& rhs)
{
    double a[4],b[4];
    double res[4];
    double *dev_a, *dev_b, *dev_c;
    hipMalloc((void**)&dev_a, 4 * sizeof(double));
    hipMalloc((void**)&dev_b, 4 * sizeof(double));
    hipMalloc((void**)&dev_c, 4 * sizeof(double));
    for (auto i = 0; i < 4; i++) {
        a[i] = this->mData[i];
        b[i] = rhs[i];
    }
    hipMemcpy(dev_a, a, 4 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, 4 * sizeof(double), hipMemcpyHostToDevice);

    dVectorAdd<<<4,1>>>(dev_a, dev_b, dev_c, 4);
    hipMemcpy(res, dev_c, 4 * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    return Vector4dCu(res);
}

SAIL_NAMESPACE_END